#include "hip/hip_runtime.h"

/* 
* Cuda kernels that does the heavy work
*/


#include "RTOneW/core.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
extern "C" void check_cuda(hipError_t result, char const *const func, const char *const file, int const line);


__device__ vec3 color(const ray& r, hitable **world, atmosphere **sky, hiprandState *local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation(1.0, 1.0, 1.0);
	vec3 cur_emmission(0, 0, 0);
	vec3 color(0, 0, 0);

	for (int i = 0; i < 50; i++) {
		hit_record rec;
		ray scattered;
		vec3 attenuation;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
			
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else {
				return emitted * attenuation;
				
			}
		}
		else {
			float t0, t1, tmax = FLT_MAX;
			vec3 orig = vec3(cur_ray.origin().x(), cur_ray.origin().y()+ (*sky)->earthRadius+2000, cur_ray.origin().z());
			if (raySphereIntersect(orig, unit_vector(cur_ray.direction()), (*sky)->earthRadius, t0, t1) && t1 > 0) tmax = ffmax(0.0f, t0);
			vec3 sky_color;
			/*if (t0 < 0)*/ sky_color = (*sky)->computeIncidentLight(orig, unit_vector(cur_ray.direction()), 0, tmax);
			//else sky_color =  vec3(.004, .002, 0);
			sky_color = vec3(0, 0, 0);
			return cur_attenuation * sky_color;
			
		}
		
	}
	return color; // exceeded recursion
}


__global__ void rand_init_kernel(hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void init_fb_kernel(vec3* fb, int max_x, int max_y) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;

	fb[pixel_index] = vec3(0, 0, 0);
}

__global__ void render_init_kernel(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render_image_kernel(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, atmosphere **sky, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);

	float u = float(i + vanDerCorput(&local_rand_state)) / float(max_x);
	float v = float(j + vanDerCorput(&local_rand_state,3)) / float(max_y);
	ray r = (*cam)->get_ray(u, v, &local_rand_state);
	col = color(r, world,sky, &local_rand_state);

	rand_state[pixel_index] = local_rand_state;
	//col /= float(ns);
	col[0] = col[0] < 1.413f ? pow(col[0] * 1.38317f, 1.0f / 2.2f) : 1.0f - exp(-col[0]);
	col[1] = col[1] < 1.413f ? pow(col[1] * 1.38317f, 1.0f / 2.2f) : 1.0f - exp(-col[1]);
	col[2] = col[2] < 1.413f ? pow(col[2] * 1.38317f, 1.0f / 2.2f) : 1.0f - exp(-col[2]);
	
	fb[pixel_index] += col;
}


__global__ void create_world_kernel(hitable **d_list, hitable **d_world, camera **d_camera, atmosphere **d_atmosphere,int nx, int ny, float fov, float aperture, hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprandState local_rand_state = *rand_state;

		constant_texture *red_texture = new constant_texture(vec3(1, 0,0));
		constant_texture *white_texture = new constant_texture(vec3(1, 1, 1));
		constant_texture *light_texture = new constant_texture(vec3(15, 15, 15));
		constant_texture *turqoise_texture = new constant_texture(vec3(0, 0.9, 0.9));
		constant_texture *green_texture = new constant_texture(vec3(0, 0.9, 0.0));
		checker_texture *checker_tex = new checker_texture(red_texture, green_texture);

		diffuse_light *light = new diffuse_light(checker_tex, 1);
		diffuse_light *white_light = new diffuse_light(light_texture, 10);
		lambertian *white_lamb = new lambertian(white_texture);

		d_list[0] = new flip_normals( new yz_rect(0,555,0,555,555,new lambertian(green_texture)));
		d_list[1] = new yz_rect(0, 555, 0, 555, 0, new lambertian(red_texture));
		d_list[2] = new xz_rect(213, 343, 227, 332, 554, white_light);
		d_list[3] = new flip_normals(new xz_rect(0, 555, 0, 555, 555, white_lamb));
		d_list[4] = new xz_rect(0, 555, 0, 555, 0, white_lamb);
		d_list[5] = new flip_normals(new xy_rect(0, 555, 0, 555, 555, white_lamb));

		*d_world = new hitable_list(d_list, 6);

		*rand_state = local_rand_state;

		vec3 lookfrom(278, 278, -800);
		vec3 lookat(278, 278, 0);
		float dist_to_focus = (lookfrom - lookat).length();

		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			fov,
			float(nx) / float(ny),
			aperture,
			dist_to_focus, 0.0f, 1.0f);

		*d_atmosphere = new atmosphere(vec3(0, 1, 0));
	}
}

__global__ void free_world_kernel(hitable **d_list, hitable **d_world, camera **d_camera) {
	for (int i = 0; i < 5; i++) {
		delete ((sphere *)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}



////////////////////////////////////////////////////////////////////////////////////////
//////// WRAPPER FUNCTIONS


extern "C" void init_fb(vec3* fb, int nx, int ny) {

	init_fb_kernel <<< 1, 1 >>> (fb, nx, ny);

}

extern "C" void rand_init(hiprandState *rand_state) {

	rand_init_kernel << <1, 1 >> > (rand_state);

}




extern "C" void render_init(int nx, int ny, int tx, int ty, hiprandState *rand_state) {

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render_init_kernel << <blocks, threads >> > (nx, ny, rand_state);


}



extern "C" void render_image(vec3 *fb, int nx, int ny, int tx, int ty, int ns, camera **cam, hitable **world, atmosphere **sky,hiprandState *rand_state) {
	
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	render_image_kernel << <blocks, threads >> > (fb, nx, ny, ns, cam, world, sky, rand_state);


}


extern "C" void create_world(hitable **d_list, hitable **d_world, camera **d_camera, atmosphere **d_atmosphere, int nx, int ny, float fov, float aperture, hiprandState *rand_state) {

	create_world_kernel <<<1, 1 >>> (d_list, d_world, d_camera, d_atmosphere ,nx, ny, fov, aperture, rand_state);

}

extern "C" void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {

	free_world_kernel <<<1, 1 >>> (d_list, d_world, d_camera);

}