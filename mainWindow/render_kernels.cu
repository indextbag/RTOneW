#include "hip/hip_runtime.h"

/* 
* Cuda kernels that does the heavy work
*/


#include "RTOneW/core.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
extern "C" void check_cuda(hipError_t result, char const *const func, const char *const file, int const line);


__device__ vec3 color(const ray& r, hitable **world, atmosphere **sky, hiprandState *local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
	
	for (int i = 0; i < 50; i++) {
		hit_record rec;
		ray scattered;
		vec3 attenuation;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			
			if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else {
				return vec3(0.0, 0.0, 0.0);
			}
		}
		else {
			float t0, t1, tmax = FLT_MAX;
			vec3 orig = vec3(cur_ray.origin().x(), cur_ray.origin().y()+ (*sky)->earthRadius+2000, cur_ray.origin().z());
			if (raySphereIntersect(orig, unit_vector(cur_ray.direction()), (*sky)->earthRadius, t0, t1) && t1 > 0) tmax = ffmax(0.0f, t0);
			vec3 sky_color;
			/*if (t0 < 0)*/ sky_color = (*sky)->computeIncidentLight(orig, unit_vector(cur_ray.direction()), 0, tmax);
			//else sky_color =  vec3(.004, .002, 0);
			return cur_attenuation * sky_color;
		}
	}
	return vec3(0.0, 0.0, 0.0); // exceeded recursion
}


__global__ void rand_init_kernel(hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprand_init(1984, 0, 0, rand_state);
	}
}

__global__ void init_fb_kernel(vec3* fb, int max_x, int max_y) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;

	fb[pixel_index] = vec3(0, 0, 0);
}

__global__ void render_init_kernel(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render_image_kernel(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, atmosphere **sky, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j * max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0, 0, 0);

	float u = float(i + vanDerCorput(&local_rand_state)) / float(max_x);
	float v = float(j + vanDerCorput(&local_rand_state,3)) / float(max_y);
	ray r = (*cam)->get_ray(u, v, &local_rand_state);
	col = color(r, world,sky, &local_rand_state);

	rand_state[pixel_index] = local_rand_state;
	//col /= float(ns);
	col[0] = col[0] < 1.413f ? pow(col[0] * 1.38317f, 1.0f / 2.2f) : 1.0f - exp(-col[0]);
	col[1] = col[1] < 1.413f ? pow(col[1] * 1.38317f, 1.0f / 2.2f) : 1.0f - exp(-col[1]);
	col[2] = col[2] < 1.413f ? pow(col[2] * 1.38317f, 1.0f / 2.2f) : 1.0f - exp(-col[2]);
	
	fb[pixel_index] += col;
}


__global__ void create_world_kernel(hitable **d_list, hitable **d_world, camera **d_camera, atmosphere **d_atmosphere,int nx, int ny, float fov, float aperture, hiprandState *rand_state) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprandState local_rand_state = *rand_state;

		constant_texture *red_texture = new constant_texture(vec3(0.9, 0,0));
		constant_texture *turqoise_texture = new constant_texture(vec3(0, 0.9, 0.9));

		d_list[0] = new sphere(vec3(0,0, 0), 0.5f, new lambertian(red_texture));
		d_list[1] = new sphere(vec3(0, -100.5, -1), 100.0f, new lambertian(turqoise_texture));
		d_list[2] = new sphere(vec3(0, 0, -1), 0.5f, new metal(vec3(1, 1, 1),0));
		d_list[3] = new sphere(vec3(0, 0, 1), 0.5f, new dielectric(1.333));

		*d_world = new hitable_list(d_list, 4);

		*rand_state = local_rand_state;

		vec3 lookfrom(-4, 0, 4);
		vec3 lookat(0, 0, 0);
		float dist_to_focus = (lookfrom - lookat).length();

		*d_camera = new camera(lookfrom,
			lookat,
			vec3(0, 1, 0),
			fov,
			float(nx) / float(ny),
			aperture,
			dist_to_focus, 0.0f, 1.0f);

		*d_atmosphere = new atmosphere(vec3(0, 1, 0));
	}
}

__global__ void free_world_kernel(hitable **d_list, hitable **d_world, camera **d_camera) {
	for (int i = 0; i < 4; i++) {
		delete ((sphere *)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}



////////////////////////////////////////////////////////////////////////////////////////
//////// WRAPPER FUNCTIONS


extern "C" void init_fb(vec3* fb, int nx, int ny) {

	init_fb_kernel <<< 1, 1 >>> (fb, nx, ny);

}

extern "C" void rand_init(hiprandState *rand_state) {

	rand_init_kernel << <1, 1 >> > (rand_state);

}




extern "C" void render_init(int nx, int ny, int tx, int ty, hiprandState *rand_state) {

	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);
	render_init_kernel << <blocks, threads >> > (nx, ny, rand_state);


}



extern "C" void render_image(vec3 *fb, int nx, int ny, int tx, int ty, int ns, camera **cam, hitable **world, atmosphere **sky,hiprandState *rand_state) {
	
	dim3 blocks(nx / tx + 1, ny / ty + 1);
	dim3 threads(tx, ty);

	render_image_kernel << <blocks, threads >> > (fb, nx, ny, ns, cam, world, sky, rand_state);


}


extern "C" void create_world(hitable **d_list, hitable **d_world, camera **d_camera, atmosphere **d_atmosphere, int nx, int ny, float fov, float aperture, hiprandState *rand_state) {

	create_world_kernel <<<1, 1 >>> (d_list, d_world, d_camera, d_atmosphere ,nx, ny, fov, aperture, rand_state);

}

extern "C" void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {

	free_world_kernel <<<1, 1 >>> (d_list, d_world, d_camera);

}